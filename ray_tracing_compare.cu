#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <string>
#include <sstream>

#include "EasyBMP.h"
#include "types.h"

using namespace std;

static void generate_scene(t_sphere * spheres, int n_spheres, t_light * lights, int n_lights)
{
	int n_random_coord = n_spheres * 3  + n_lights * 3;
	int n_random_rad = n_spheres;
	int n_random_colors = n_spheres * 3;

	size_t n = n_random_coord + n_random_rad + n_random_colors;

	hiprandGenerator_t gen;
	float *devData, *hostData;
	hostData = (float *)calloc(n, sizeof(float));

	if (!hostData)
	{
		fprintf(stderr, "Malloc error, exiting\n");
		exit(-1);
	}

	CUDA_CALL( hipMalloc((void **)&devData, n*sizeof(float)) );

	CURAND_CALL( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
	CURAND_CALL( hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)time(NULL)) ); 

	CURAND_CALL( hiprandGenerateUniform(gen, devData, n) );
	CUDA_CALL( hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost) );

	float x_pos = 0.9f;
	float y_pos = BOX_SIZE / 5.0;

	for (int i = 0; i < n_spheres; i++)
	{
		spheres[i].center.x = x_pos;
		spheres[i].center.y = y_pos;

		x_pos += BOX_SIZE / (n_spheres / 2.0);

		if  ( x_pos > BOX_SIZE - 0.9)
		{
			x_pos = 0.9f;
			y_pos = BOX_SIZE / 2.5 ;
		}
	}
	
	int j = 0;
	for (int i = 0; i < n_spheres; i++)
	{
		spheres[i].center.x += 2.0 * (hostData[j++] - 0.5);
		spheres[i].center.y += 2.0 * (hostData[j++] - 0.5);
		spheres[i].center.z = hostData[j++] * BOX_SIZE_Z + DISTANCE;
		spheres[i].radius = hostData[j++] * RADIUS_MAX + RADIUS_MIN;
		spheres[i].red   = hostData[j++] / (DEPTH_MAX - 3);
		spheres[i].green = hostData[j++] / (DEPTH_MAX - 3);
		spheres[i].blue  = hostData[j++] / (DEPTH_MAX - 3);
	}

	for (int i = 0; i < n_lights; i++)
	{
		lights[i].x = (hostData[j++] - 0.5) * BOX_SIZE * 6;
		lights[i].y = (hostData[j++] - 0.5) * BOX_SIZE * 6;
		lights[i].z = hostData[j++] * DISTANCE/2.0;
	}

	CURAND_CALL( hiprandDestroyGenerator(gen) );
	CUDA_CALL( hipFree(devData) );
	free(hostData);    
}

int main(int argc, char** argv)
{
	if (argc != 6)
	{
		printf("Usage: %s <n_spheres> <n_lights> <width> <height> <bmp_filename> \n", argv[0]);
		return -1;
	}

	int n_spheres = atoi(argv[1]);
	int n_lights = atoi(argv[2]);
	int width = atoi(argv[3]);
	int height = atoi(argv[4]);
	char* filename = argv[5];

	// Generate random scene config and store it in the command line args
	t_sphere * spheres = (t_sphere *) malloc (sizeof(t_sphere) * n_spheres);
	t_light * lights = (t_light *) malloc (sizeof(t_light) * n_lights);

	if (lights == NULL || spheres == NULL)
	{
		fprintf(stderr, "Malloc error, exiting\n");
		exit(-1);
	}

	generate_scene(spheres, n_spheres, lights, n_lights);
	
	string args;
	{
		stringstream s;
		s << n_spheres << " ";
		for (int i = 0; i < n_spheres; i++)
		{
			s << spheres[i].center.x << " ";
			s << spheres[i].center.y << " ";
			s << spheres[i].center.z << " ";
			s << spheres[i].radius << " ";
			s << spheres[i].red << " ";
			s << spheres[i].green << " ";
			s << spheres[i].blue << " ";
		}
		s << n_lights << " ";
		for (int i = 0; i < n_lights; i++)
		{
			s << lights[i].x << " "; 
			s << lights[i].y << " "; 
			s << lights[i].z << " ";; 
		}
		s << width << " " << height << " " << filename;
		args = s.str();
	}

	// Launch CUDA ray tracer
	{
		stringstream s;
		s << "cd cuda && ./ray_tracing_cuda " << args;
		string cmd = s.str();
		//cout << cmd << endl;
		int result = system(cmd.c_str());
		if (result) return result;
	}

	// Launch OptiX ray tracer
	{
		stringstream s;
		s << "cd optix && ./ray_tracing_optix " << args;
		string cmd = s.str();
		//cout << cmd << endl;
		int result = system(cmd.c_str());
		if (result) return result;
	}	

	// Compare BMP outputs
	BMP ImageCUDA;
	ImageCUDA.ReadFromFile("cuda/output.bmp");
	BMP ImageOptix;
	ImageOptix.ReadFromFile("optix/output.bmp");
	width = ImageCUDA.TellWidth();
	height = ImageCUDA.TellHeight();
	if (width != ImageOptix.TellWidth())
	{
		fprintf(stderr, "CUDA and Optix output images widths mismatch: %d != %d",
			width, ImageOptix.TellWidth());
		return 1;
	}
	if (height != ImageOptix.TellHeight())
	{
		fprintf(stderr, "CUDA and Optix output images heights mismatch: %d != %d",
			height, ImageOptix.TellHeight());
		return 1;
	}
	
	uint maxRdiff = 0, maxGdiff = 0, maxBdiff = 0;
	float avgRdiff = 0.0, avgGdiff = 0.0, avgBdiff = 0.0;
	for (int i = 0; i < width; i++)
	{
		for (int j = 0; j < height; j++)
		{
			RGBApixel PixelCUDA = ImageCUDA.GetPixel(i, j);
			RGBApixel PixelOptix = ImageOptix.GetPixel(i, j);
			
			maxRdiff = max(maxRdiff, abs(PixelCUDA.Red - PixelOptix.Red));
			maxGdiff = max(maxGdiff, abs(PixelCUDA.Green - PixelOptix.Green));
			maxBdiff = max(maxBdiff, abs(PixelCUDA.Blue - PixelOptix.Blue));

			avgRdiff += abs(PixelCUDA.Red - PixelOptix.Red);
			avgGdiff += abs(PixelCUDA.Green - PixelOptix.Green);
			avgBdiff += abs(PixelCUDA.Blue - PixelOptix.Blue);
		}
	}

	printf("Images max abs difference: { R, G, B } = { %u, %u, %u }\n",
		(uint)maxRdiff, (uint)maxGdiff, (uint)maxBdiff);
	printf("Images average difference: { R, G, B } = { %f, %f, %f }\n",
		avgRdiff / (width * height), avgGdiff / (width * height), avgBdiff / (width * height));

	return 0;
}

