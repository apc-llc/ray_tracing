#include <cstdio>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <string>
#include <sstream>

#include "types.h"

using namespace std;

static void generate_scene(t_sphere * spheres, int n_spheres, t_light * lights, int n_lights)
{
	int n_random_coord = n_spheres * 3  + n_lights * 3;
	int n_random_rad = n_spheres;
	int n_random_colors = n_spheres * 3;

	size_t n = n_random_coord + n_random_rad + n_random_colors;

	hiprandGenerator_t gen;
	float *devData, *hostData;
	hostData = (float *)calloc(n, sizeof(float));

	if (!hostData)
	{
		fprintf(stderr, "Malloc error, exiting\n");
		exit(-1);
	}

	CUDA_CALL( hipMalloc((void **)&devData, n*sizeof(float)) );

	CURAND_CALL( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
	CURAND_CALL( hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)time(NULL)) ); 

	CURAND_CALL( hiprandGenerateUniform(gen, devData, n) );
	CUDA_CALL( hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost) );
	
	int j = 0;
	for (int i = 0; i < n_spheres; i++)
	{
		spheres[i].center.x = hostData[j++] * BOX_SIZE ;
		spheres[i].center.y = hostData[j++] * BOX_SIZE ;
		spheres[i].center.z = hostData[j++] * BOX_SIZE + DISTANCE ;
		spheres[i].radius = hostData[j++] * RADIUS_MAX + RADIUS_MIN;
		spheres[i].red   = hostData[j++] / (DEPTH_MAX - 3);
		spheres[i].green = hostData[j++] / (DEPTH_MAX - 3);
		spheres[i].blue  = hostData[j++] / (DEPTH_MAX - 3);
	}

	for (int i = 0; i < n_lights; i++)
	{
		lights[i].x = hostData[j++] * BOX_SIZE; 
		lights[i].y = hostData[j++] * BOX_SIZE; 
		lights[i].z = hostData[j++] * DISTANCE + BOX_SIZE / 2.0; 
	}

	CURAND_CALL( hiprandDestroyGenerator(gen) );
	CUDA_CALL( hipFree(devData) );
	free(hostData);    
}

int main(int argc, char** argv)
{
	if (argc != 6)
	{
		printf("Usage: %s <n_spheres> <n_lights> <width> <height> <bmp_filename> \n", argv[0]);
		return -1;
	}

	int n_spheres = atoi(argv[1]);
	int n_lights = atoi(argv[2]);
	int width = atoi(argv[3]);
	int height = atoi(argv[4]);
	char* filename = argv[5];

	// Generate random scene config and store it in the command line args
	t_sphere * spheres = (t_sphere *) malloc (sizeof(t_sphere) * n_spheres);
	t_light * lights = (t_light *) malloc (sizeof(t_light) * n_lights);

	if (lights == NULL || spheres == NULL)
	{
		fprintf(stderr, "Malloc error, exiting\n");
		exit(-1);
	}

	generate_scene(spheres, n_spheres, lights, n_lights);
	
	string args;
	{
		stringstream s;
		s << n_spheres << " ";
		for (int i = 0; i < n_spheres; i++)
		{
			s << spheres[i].center.x << " ";
			s << spheres[i].center.y << " ";
			s << spheres[i].center.z << " ";
			s << spheres[i].radius << " ";
			s << spheres[i].red << " ";
			s << spheres[i].green << " ";
			s << spheres[i].blue << " ";
		}
		s << n_lights << " ";
		for (int i = 0; i < n_lights; i++)
		{
			s << lights[i].x << " "; 
			s << lights[i].y << " "; 
			s << lights[i].z << " ";; 
		}
		s << width << " " << height << " " << filename;
		args = s.str();
	}

	// Launch CUDA ray tracer
	{
		stringstream s;
		s << "cd cuda && ./ray_tracing_cuda " << args;
		string cmd = s.str();
		cout << cmd << endl;
		int result = system(cmd.c_str());
		if (result) return result;
	}

	// Launch OptiX ray tracer
	{
		stringstream s;
		s << "cd optix && ./ray_tracing_optix " << args;
		string cmd = s.str();
		cout << cmd << endl;
		int result = system(cmd.c_str());
		if (result) return result;
	}	

	// Compare BMP outputs

	return 0;
}

