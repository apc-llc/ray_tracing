#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include <math.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "debug.h"
#include "ray_tracing.h"

#include "kernel.h"


#define CUDA_CALL(x) do { hipError_t err = x; if (( err ) != hipSuccess ) { \
	printf ("Error \"%s\" at %s :%d \n" , hipGetErrorString(err), \
			__FILE__ , __LINE__ ) ; exit(-1);\
}} while (0)

#define CURAND_CALL(x) do { if (( x ) != HIPRAND_STATUS_SUCCESS ) {\
	printf ("Error at %s :%d \n" , __FILE__ , __LINE__ ) ;\
	exit(-1); }} while (0)




//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b)
{
    return (a % b != 0) ? (a - a % b + b) : a;
}


//#define INITIALIZE_HACK


void generate_scene(t_sphere * spheres, int n_spheres, t_light * lights, int n_lights)
{
	int n_random_coord = n_spheres * 3  + n_lights * 3;
	int n_random_rad = n_spheres;
	int n_random_colors = n_spheres * 3;

	size_t n = n_random_coord + n_random_rad + n_random_colors;

	hiprandGenerator_t gen;
	float *devData, *hostData;
	hostData = (float *)calloc(n, sizeof(float));
	CUDA_CALL( hipMalloc((void **)&devData, n*sizeof(float)) );

	CURAND_CALL( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
	CURAND_CALL( hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)time(NULL)) ); 

	CURAND_CALL( hiprandGenerateUniform(gen, devData, n) );
	CUDA_CALL( hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost) );
	
	int j=0;
	for (int i=0; i<n_spheres; i++)
	{
		spheres[i].center.x = hostData[j + 0] * BOX_SIZE ;
		spheres[i].center.y = hostData[j + 1] * BOX_SIZE ;
		spheres[i].center.z = hostData[j + 2] * BOX_SIZE + DISTANCE ;
		spheres[i].radius = hostData[j + 3] * RADIUS_MAX + RADIUS_MIN;

		spheres[i].red   = hostData[j + 4] / (DEPTH_MAX-3);
		spheres[i].green = hostData[j + 5] / (DEPTH_MAX-3);
		spheres[i].blue  = hostData[j + 6] / (DEPTH_MAX-3);
		j+=7;
	}

	for (int i=0; i<n_lights; i++)
	{
		lights[i].x = hostData[j + 0] * BOX_SIZE; 
		lights[i].y = hostData[j + 1] * BOX_SIZE; 
		lights[i].z = hostData[j + 2] * DISTANCE + BOX_SIZE/2.0; 
		j+=3; 
	}



#ifdef INITIALIZE_HACK
	spheres[0].center.x=9.746 ;
	spheres[0].center.y=7.0 ;
	spheres[0].center.z=29.0 ;
	spheres[0].radius=0.815 ;
	spheres[0].red=0.683 ;
	spheres[0].green=0.133 ;
	spheres[0].blue=0.403  ;

	spheres[1].center.x=3.372 ;
	spheres[1].center.y=3.0 ;
	spheres[1].center.z=29.0 ;
	spheres[1].radius=2.810 ;
	spheres[1].red=0.305 ;
	spheres[1].green=0.156 ;
	spheres[1].blue=0.199  ;

	lights[0].x=15.878 ;
	lights[0].y=3.0 ;
	lights[0].z=29.0 ;

//	lights[1].x=6.518 ;
//	lights[1].y=6.930 ;
//	lights[1].z=31.367 ;
#endif



	CURAND_CALL( hiprandDestroyGenerator(gen) );
	CUDA_CALL( hipFree(devData) );
	free(hostData);    
}




void ray_trace(unsigned char * pR, unsigned char * pG, unsigned char * pB, 
				int height, int width, int n_spheres, int n_lights)
{
	hipEvent_t start=0, stop=0;
        float gpuTime = 0.0f;

	hipError_t err;

//#define STACK_INCREASE
#ifdef STACK_INCREASE 
	size_t stack=0;
	CUDA_CALL( hipDeviceGetLimit(&stack, hipLimitStackSize) ); 
	printf ("Cuda stack size %ld \n", stack);
	stack = 1536;
	printf ("Setting cuda stack size to %ld \n", stack);
	CUDA_CALL( hipDeviceSetLimit(hipLimitStackSize, stack) );
#endif

	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	//hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	CUDA_CALL (hipEventCreate (&start) );
        CUDA_CALL (hipEventCreate (&stop) );

	t_sphere * spheres = (t_sphere *) malloc (sizeof(t_sphere) * n_spheres);
	t_light * lights = (t_light *) malloc (sizeof(t_light) * n_lights);

	generate_scene(spheres, n_spheres, lights, n_lights);



#ifdef DEBUG
	print_spheres(spheres, n_spheres);
	print_lights(lights, n_lights);
#endif

	t_sphere * dev_spheres;
	t_light * dev_lights;

	CUDA_CALL( hipMalloc((void **)&dev_spheres,  sizeof(t_sphere) * n_spheres ) );
	CUDA_CALL( hipMalloc((void **)&dev_lights,  sizeof(t_light) * n_lights ) );

	CUDA_CALL( hipMemcpy(dev_spheres, spheres, sizeof(t_sphere) * n_spheres, hipMemcpyHostToDevice) );
	CUDA_CALL( hipMemcpy(dev_lights, lights, sizeof(t_light) * n_lights, hipMemcpyHostToDevice) );

	unsigned char * dev_image_red;
	unsigned char * dev_image_green;
	unsigned char * dev_image_blue;

	CUDA_CALL( hipMalloc((void **)&dev_image_red,   height * width *sizeof(unsigned char)) );
	CUDA_CALL( hipMalloc((void **)&dev_image_green, height * width *sizeof(unsigned char)) );
	CUDA_CALL( hipMalloc((void **)&dev_image_blue,  height * width *sizeof(unsigned char)) );

	CUDA_CALL( hipMemset(dev_image_red,   0, height * width *sizeof(unsigned char)) );
	CUDA_CALL( hipMemset(dev_image_green, 0, height * width *sizeof(unsigned char)) );
	CUDA_CALL( hipMemset(dev_image_blue,  0, height * width *sizeof(unsigned char)) );

	CUDA_CALL( hipEventRecord (start, 0) );

	dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
	dim3 grid(iDivUp(width, block.x), iDivUp(height, block.y), 1);

#ifdef DEBUG
	printf ("Running kernel with block.x=%d block.y=%d \n", block.x, block.y);
	printf ("Running kernel with grid.x=%d grid.y=%d \n", grid.x, grid.y);
#endif

	kernel<<<grid,block>>>(dev_image_red, dev_image_blue, dev_image_green, 
			height, width, dev_spheres, n_spheres, dev_lights, n_lights); 

	err = hipGetLastError(); 
	if (err != hipSuccess) 
	{
		printf( "%s \n", hipGetErrorString( err ) );
	}

	CUDA_CALL( hipMemcpy(pR, dev_image_red,  height * width *sizeof(unsigned char), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipMemcpy(pB, dev_image_blue, height * width *sizeof(unsigned char), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipMemcpy(pG, dev_image_green,height * width *sizeof(unsigned char), hipMemcpyDeviceToHost) );

	CUDA_CALL( hipEventRecord (stop, 0) );
        CUDA_CALL( hipEventSynchronize(stop) );
        CUDA_CALL( hipEventElapsedTime (&gpuTime, start, stop) );

	printf("GPU ray tracing  \n");
        printf("time spent executing on GPU: %.2f milliseconds\n", gpuTime);

	CUDA_CALL( hipFree(dev_image_red) );
	CUDA_CALL( hipFree(dev_image_green) );
	CUDA_CALL( hipFree(dev_image_blue) );

	CUDA_CALL( hipFree(dev_spheres) );
	CUDA_CALL( hipFree(dev_lights) );

	CUDA_CALL( hipEventDestroy (start) );
	CUDA_CALL( hipEventDestroy (stop) );

	free (spheres);
	free (lights);

	return ;
}


