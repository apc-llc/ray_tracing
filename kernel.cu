#include "hip/hip_runtime.h"
#include "kernel.h"
#include <float.h>
#include <stdio.h>


#define X_DIM BOX_SIZE

__device__ __shared__ t_vector view_point;
__device__ __shared__ float y_dim;

__device__ __shared__ int n_spheres; 
__device__ __shared__ int n_lights;
__device__ __shared__ t_sphere spheres[ SPHERES_MAX ];
__device__ __shared__ t_light  lights[ LIGHTS_MAX  ];


#define EPSILON 1e-2
__device__  bool fequal(float a, float b)
{
	return fabs( __fadd_rn(a, -b) ) < EPSILON;
}

__device__ void vec_sub (t_vector *v1, t_vector *v2, t_vector *v3) {

	v1->x = __fadd_rn( v2->x, -v3->x);
	v1->y = __fadd_rn( v2->y, -v3->y);
	v1->z = __fadd_rn( v2->z, -v3->z);
}

__device__ void vec_add (t_vector *v1, t_vector *v2, t_vector *v3) {
	
	v1->x = __fadd_rn (v2->x, v3->x);
	v1->y = __fadd_rn (v2->y, v3->y);
	v1->z = __fadd_rn (v2->z, v3->z);
}

__device__ void vec_scale (float scale, t_vector *v1, t_vector *v2) {
	
	v1->x = __fmul_rn (scale, v2->x); // multiplying
	v1->y = __fmul_rn (scale, v2->y);
	v1->z = __fmul_rn (scale, v2->z);
}

__device__ float dotproduct (t_vector *v1, t_vector *v2) {
	
	return 
		__fadd_rn(
		 	__fmul_rn (v1->x, v2->x), 
			__fadd_rn ( __fmul_rn (v1->y, v2->y),  __fmul_rn (v1->z, v2->z))
		 );
}


__device__ void normalize_vector (t_vector *v) {
	
	float magnitude;
	
	magnitude = __fsqrt_rn ( dotproduct(v, v) );
	v->x = __fdiv_rn (v->x, magnitude);
	v->y = __fdiv_rn (v->y, magnitude);
	v->z = __fdiv_rn (v->z, magnitude);
}

__device__ void compute_ray(t_ray* ray, t_vector* view_point,
			 t_pixel* pixel) 
{
	ray->origin = *view_point;

	ray->direction.x = 
		__fdiv_rn (__fmul_rn (X_DIM, pixel->i), 
			__mul24(blockDim.x, gridDim.x)) - __fdiv_rn (X_DIM, 2.0) ;

	ray->direction.y = 
		__fdiv_rn (__fmul_rn (y_dim, pixel->j), 
			__mul24(blockDim.y, gridDim.y)) - __fdiv_rn (y_dim, 2.0) ;

	ray->direction.z = (float) DISTANCE;

	normalize_vector(&ray->direction);
}


__device__ void compute_reflected_ray(t_ray* reflected_ray, t_ray* incidence_ray, 
		t_sphere_intersection* intersection) 
{
	
	float dp1;
	t_vector scaled_normal;
	reflected_ray->origin=intersection->point;

	dp1 = dotproduct(&intersection->normal, &incidence_ray->direction);
	dp1 = __fmul_rn (2, dp1);

	vec_scale(dp1, &scaled_normal, &intersection->normal);
	
	vec_sub(&reflected_ray->direction, &incidence_ray->direction, &scaled_normal);
}


__device__ void compute_ray_to_light(t_ray* ray, 
		t_sphere_intersection* intersection, t_vector* light)
{
	ray->origin = intersection->point;
        vec_sub(&ray->direction, light, &intersection->point);
        normalize_vector(&ray->direction);
}


__device__ bool sphere_intersection (t_ray *ray, t_sphere *sphere, 
		t_sphere_intersection* intersection) 
{

	float discriminant;
	float A, B, C;
	float lambda1, lambda2;
	t_vector temp;
	
	A = dotproduct(&ray->direction, &ray->direction);
	
	vec_sub(&temp, &ray->origin, &sphere->center);
	B = __fmul_rn (2.0, dotproduct(&temp, &ray->direction));
	
	C = __fadd_rn( dotproduct(&temp, &temp), 
		-__fmul_rn( sphere->radius, sphere->radius ));
	
	discriminant = __fadd_rn( __fmul_rn(B, B), 
		-__fmul_rn(4.0, __fmul_rn(A, C)));
	
	if (discriminant >= 0) {
		lambda1 = __fdiv_rn (__fadd_rn(-B,  __fsqrt_rn(discriminant)), 
				__fmul_rn(2.0, A));
		lambda2 = __fdiv_rn (__fadd_rn(-B, -__fsqrt_rn(discriminant)), 
				__fmul_rn(2.0, A));

		intersection->lambda_in = fminf(lambda1, lambda2);

		// is the object visible from the eye (lambda1,2>0)
		if (fequal( intersection->lambda_in, 0.0) || (lambda1>0 && lambda2>0) ){
			return true;
		}
	}
	return false;
}


// Calculate normal vector in the point of intersection:
__device__ void intersection_normal(t_sphere *sphere, 
		t_sphere_intersection* intersection, t_ray* ray) 
{
	float  scale;
	t_vector tmp_vec;
	
	//calculating coordinates of intersection point
	vec_scale(intersection->lambda_in, &tmp_vec, &ray->direction);
	vec_add(&intersection->point, &tmp_vec, &ray->origin);

	//calculating direction of normal in the point of intersection 
	vec_sub(&tmp_vec, &intersection->point, &sphere->center);
	
	//scaling normal vector
	scale = __frcp_rn(sphere->radius);
	vec_scale(scale, &intersection->normal, &tmp_vec);
	normalize_vector(&intersection->normal);
}





 __device__ t_color TraceRay(t_ray ray, int depth )
{
	t_ray ray_tmp;
	t_color illumination={0.0, 0.0, 0.0};
	t_color tmp;

	if( depth > DEPTH_MAX )
	{
		return illumination ;
	}

	t_sphere_intersection intersection, current_intersection;
	int intersection_object = -1; // none
	int k,i;

	float visible = 1.0;
	float current_lambda = FLT_MAX; // maximum positive float
	int count=0;

	//find closest ray object / intersection ;
	for (k=0; k<n_spheres; k++)
	{
		if (sphere_intersection(&ray, &spheres[k], &intersection))
		{
			if (intersection.lambda_in<current_lambda)
			{
				current_lambda=intersection.lambda_in;
				intersection_object=k;
				current_intersection=intersection;
			}
		}
	}
	//if( intersection exists )
        if (intersection_object > -1)
        {
		intersection_normal(&spheres[intersection_object], &current_intersection, &ray);
		//for each light source in the scene
		for (i=0; i<n_lights; i++)
		{
			compute_ray_to_light(&ray_tmp, &current_intersection, &lights[i]);

			for (k=0; k<n_spheres; k++)
			{
				if (sphere_intersection
					(&ray_tmp, &spheres[k], &intersection)
				   )
				{
					if (count++ == 0)
					{   
						visible = 0.2; 
					}else
					{   
						visible = 0.0;
					}   
					break;
				}
			}


			illumination.red   = __fadd_rn (
				illumination.red, __fmul_rn(visible, spheres[intersection_object].red));
			illumination.green = __fadd_rn (
				illumination.green, __fmul_rn(visible, spheres[intersection_object].green));
			illumination.blue  = __fadd_rn (
				illumination.blue, __fmul_rn(visible, spheres[intersection_object].blue));


		}
		compute_reflected_ray(&ray_tmp, &ray, &current_intersection);

		tmp = TraceRay(ray_tmp, depth+1 );

		illumination.red   = __fadd_rn (illumination.red,  tmp.red);
		illumination.blue  = __fadd_rn (illumination.blue, tmp.blue);
		illumination.green = __fadd_rn (illumination.green,tmp.green);

	}
	
	return illumination;
}





__global__ void kernel(unsigned char * dev_image_red, 
			unsigned char * dev_image_blue, 
			unsigned char * dev_image_green, 
			int  height, int width, 
			t_sphere * dev_spheres, int dev_n_spheres, 
			t_light * dev_lights, int dev_n_lights)
{

	t_color illumination;
	t_ray ray;
	t_pixel pixel;

	pixel.i = blockIdx.x * blockDim.x + threadIdx.x; // x coordinate inside whole picture
	pixel.j = blockIdx.y * blockDim.y + threadIdx.y; // y coordinate inside whole picture

	if (pixel.i>= width || pixel.j>=height)
	{
		return;
	}
	 
	int idx = threadIdx.x + threadIdx.y * blockDim.x; //linear index inside a block

	// is there a way to overcome warp divergence?
	if (threadIdx.x ==0 && threadIdx.y==0)
	{
		n_spheres = dev_n_spheres;
		n_lights = dev_n_lights;

		y_dim = __fdiv_rn (BOX_SIZE, __fdiv_rn ( (float) width, (float) height ));

		view_point.x = __fdiv_rn (X_DIM, 2.0);
		view_point.y = __fdiv_rn (y_dim, 2.0); 
		view_point.z = 0; 
	}

//	if (threadIdx.x < n_spheres && threadIdx.y==0 )
//	{
//		spheres[threadIdx.x].center = dev_spheres[threadIdx.x].center;
//		spheres[threadIdx.x].radius = dev_spheres[threadIdx.x].radius;
//		spheres[threadIdx.x].red    = dev_spheres[threadIdx.x].red;
//		spheres[threadIdx.x].green  = dev_spheres[threadIdx.x].green;
//		spheres[threadIdx.x].blue   = dev_spheres[threadIdx.x].blue;
//	}
//	if (threadIdx.x <n_lights && threadIdx.y==0)
//	{
//		lights[threadIdx.x] = dev_lights[threadIdx.x];
//	}

	if (idx < n_spheres * int(sizeof(t_sphere)/sizeof(float)) )
	{
		( (float * )spheres )[idx] = ((float *)dev_spheres)[idx];
	}
	__syncthreads();

	if (idx <n_lights * int(sizeof(t_light)/sizeof(float)) )
	{
		( (float * )lights )[idx] = ((float *) dev_lights)[idx];
	}
	__syncthreads();
    

	//compute ray starting point and direction ;
	compute_ray(&ray, &view_point, &pixel);
	illumination = TraceRay(ray, 0) ;
	//pixel color = illumination tone mapped to displayable range ;

	if (illumination.red>1.0)
		illumination.red=1.0;
	if (illumination.green>1.0)
		illumination.green=1.0;
	if (illumination.blue>1.0)
		illumination.blue=1.0;


	idx = pixel.i + __mul24(width, pixel.j);

	dev_image_red  [idx ]  = 
		(unsigned char) round (__fmul_rn (RGB_MAX, illumination.red));

	dev_image_green[ idx ]  = 
		(unsigned char) round (__fmul_rn (RGB_MAX, illumination.green));

	dev_image_blue [ idx ]  = 
		(unsigned char) round (__fmul_rn (RGB_MAX, illumination.blue));

}



