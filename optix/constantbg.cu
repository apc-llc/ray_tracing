#include "hip/hip_runtime.h"
#include <optix_world.h>

rtDeclareVariable(float3, bg_color, , );

struct PerRayData_radiance
{
	float3 result;
	float importance;
	int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}

